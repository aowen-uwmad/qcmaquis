#include "hip/hip_runtime.h"
//
//  kernels_gpu_interface.h
//  vli
//
//  Created by Timothée Ewart on 22/08/11.
//  Copyright 2011 Université de Genève. All rights reserved.
//

#include "vli/detail/kernels_cpu_gpu.hpp"
#include "vli/detail/kernels_gpu.h"


namespace vli {
namespace detail {

/**
* local declaration of the commun kernels
*/

template <typename BaseInt, std::size_t Size>
__host__ __device__ void kernels_addition_classic(BaseInt* x, BaseInt const* y);

template <typename BaseInt, std::size_t Size>
__host__ __device__ void kernel_negate_device(BaseInt* x);
 
template <typename BaseInt>
__host__ __device__ void kernels_addition_block(BaseInt* x, BaseInt const* y); 

template <typename BaseInt, std::size_t Size>
__host__ __device__ void kernels_multiplication_classic_truncate(BaseInt * res, BaseInt const* x, BaseInt const* y);	

template <typename BaseInt>
__host__ __device__ void kernels_multiplication_block(BaseInt const* x, BaseInt const* y, BaseInt* r);

template <typename BaseInt>
__host__ __device__ void kernels_multiplication_block_down(BaseInt const* x, BaseInt const*  y, BaseInt* r);

template <typename BaseInt>
__host__ __device__ void kernels_multiplication_block_up(BaseInt const* x, BaseInt const*  y, BaseInt* r);	

template <typename BaseInt>
__host__ __device__ void kernels_multiplication_base_reshaping(BaseInt const* x, BaseInt  const*  y, BaseInt* r);	

template <typename BaseInt, std::size_t Size>
__device__ void single_multiplication_device(BaseInt const* x, BaseInt const* y, BaseInt* z);  

template <typename BaseInt, std::size_t Size>
__device__ void polynome_polynome_multiplication_device(BaseInt const* p1, BaseInt const* p2, BaseInt* res);


/**
* a kind of hook function with a little bit of arithmetic in case of signed int (multiplication)
*/

/**
* VLI_GPU functions
*/

template <typename BaseInt, std::size_t Size>
__device__  void single_multiplication_device(BaseInt const* x, BaseInt const* y, BaseInt* z)  
{
 //   int na(1),nb(1);

    bool result_is_negative = static_cast<bool>((x[Size-1] ^ y[Size-1]) >> data_bits<BaseInt>::value);
    if(result_is_negative)// test if 
    {
        kernel_negate_device<BaseInt,Size>(const_cast<BaseInt* >(x));
        kernels_multiplication_classic_truncate<BaseInt,Size>(z,x,y); 
        kernel_negate_device<BaseInt,Size>(const_cast<BaseInt* >(x));
    }
    else
    {
        kernels_multiplication_classic_truncate<BaseInt,Size>(z,x,y); 
    }

/*    
    if( static_cast<bool>((x[Size-1]) >> data_bits<BaseInt>::value)){
        kernel_negate_device<BaseInt,Size>(const_cast<BaseInt* >(x));
        na = -1;
    }

    if( static_cast<bool>((y[Size-1]) >> data_bits<BaseInt>::value)){
        kernel_negate_device<BaseInt,Size>(const_cast<BaseInt* >(y));
        nb = -1;
    }

    kernels_multiplication_classic_truncate<BaseInt,Size>(z,x,y);

    if(nb*na == -1)
        kernel_negate_device<BaseInt,Size>(z);
       
    if(na == -1){
        kernel_negate_device<BaseInt,Size>(const_cast<BaseInt* >(x));
    }

    if(nb == -1){
        kernel_negate_device<BaseInt,Size>(const_cast<BaseInt* >(y));
    }*/
}
/**
I try to implement this stuff into the commun kernel impossible ... (3 days of trying)
the compiler does not want, we should move it inside ! 
**/
template <typename BaseInt, std::size_t Size>
__device__ void kernel_negate_device(BaseInt* x)
{
    BaseInt one(1);
    for(std::size_t i=0; i < Size-1; ++i)
        *(x+i) = (~*(x+i))&data_mask<BaseInt>::value;
    *(x+Size-1) = (~*(x+Size-1))&(base<BaseInt>::value+data_mask<BaseInt>::value);
    
    // TODO BUG!!!!
    kernels_addition_block(x,&one);
}


template <typename BaseInt, std::size_t Size>
__device__ void polynome_polynome_multiplication_device(unsigned int max_order, BaseInt const* p1, BaseInt const* p2, BaseInt* res)
{
    for(std::size_t je1 = 0; je1 < max_order; ++je1)
    {
        for(std::size_t je2 = 0; je2 < max_order; ++je2)
        {
            for(std::size_t he1 = 0; he1 < max_order; ++he1)
            {
                for(std::size_t he2 = 0; he2 < max_order; ++he2)
                {
                    BaseInt inter[Size];
                    #pragma unroll
                    for(std::size_t i=0 ; i < Size ;++i)
                        inter[i] = 0;

                    std::size_t offset0 = ((je1+je2)*max_order + he1+he2)*Size;
                    std::size_t offset1 = (je1*max_order+he1)*Size;
                    std::size_t offset2 = (je2*max_order+he2)*Size;

                    single_multiplication_device<BaseInt,Size>(&p1[offset1],&p2[offset2],&inter[0]);
                    kernels_addition_classic<BaseInt,Size>(&res[offset0],&inter[0]);
                } 
            }
        }      
    }
} 

/**
* VLI_GPU_VECTOR functions
*/

    
template  <typename BaseInt, std::size_t Size>
__global__ void inner_prod_vector(unsigned int max_order, std::size_t vector_size, BaseInt const* v1, BaseInt const* v2, BaseInt* res)
{
    unsigned int xIndex = blockIdx.x*blockDim.x + threadIdx.x; // all index on x
    const std::size_t size_multiplicant = Size*max_order*max_order;
    const std::size_t size_product = Size*2*max_order*2*max_order;
    //multiplication between polynomial
    if(xIndex < vector_size)
    {
        std::size_t offset_m = xIndex*size_multiplicant;
        std::size_t offset_p = xIndex*size_product;
        polynome_polynome_multiplication_device<BaseInt,Size>(max_order,&v1[offset_m],&v2[offset_m],&res[offset_p]); 
    }
}
    
    
template  <typename BaseInt, std::size_t Size>
__global__ void reduction_polynome(unsigned int max_order, std::size_t vector_size, BaseInt* v1)
{ 
    std::size_t size_poly = Size*max_order*max_order;
    for(std::size_t i=1 ; i < vector_size ; ++i){
        for(std::size_t j=0 ; j < max_order*max_order; ++j){ //additional loop
            std::size_t offset0 = j*Size;
            std::size_t offset1 = i*size_poly+j*Size;
            kernels_addition_classic<BaseInt,Size>(&v1[offset0],&v1[offset1]);
        }
    }
}
   
/**
  * The C++ functions that call the kernels
  */

template <typename BaseInt, std::size_t Size>
void inner_product_vector(unsigned int max_order, std::size_t vector_size, BaseInt const* A, BaseInt const* B, BaseInt* C, std::size_t threads_per_block) 
{
    std::size_t blocks_per_grid = vector_size/threads_per_block+1;
    dim3 dimgrid(blocks_per_grid,1,1);
    dim3 dimblock(threads_per_block,1,1);
 /* 
    std::size_t blocks_per_grid = vector_size;
    dim3 dimgrid(blocks_per_grid, 1, 1);
    dim3 dimblock(max_order, max_order, 1);
*/
    inner_prod_vector<BaseInt, Size> <<< dimgrid, dimblock >>>(max_order, vector_size, A, B, C);
} 

template <typename BaseInt, std::size_t Size>
void vector_reduction_inplace(unsigned int max_order, std::size_t vector_size, BaseInt* A)
{
    //the reduction should be // if executed on one smp
    dim3 dimgrid(1,1,1);
    dim3 dimblock(1,1,1);
    reduction_polynome<BaseInt, Size> <<< dimgrid, dimblock >>>(max_order, vector_size, A);
}

#define VLI_IMPLEMENT_GPU_FUNCTIONS(TYPE, VLI_SIZE) \
    void inner_product_vector(vli_size_tag<VLI_SIZE>, unsigned int max_order, std::size_t vector_size, TYPE const* A, TYPE const* B, TYPE* C, std::size_t threads_per_block) \
        {inner_product_vector<TYPE,VLI_SIZE>(max_order,vector_size,A,B,C,threads_per_block);} \
    void vector_reduction_inplace(vli_size_tag<VLI_SIZE>, unsigned int max_order, std::size_t vector_size, TYPE* A) \
        {vector_reduction_inplace<TYPE,VLI_SIZE>(max_order,vector_size,A);}

#define VLI_IMPLEMENT_GPU_FUNCTIONS_FOR(r, data, BASEINT_SIZE_PAIR) \
    VLI_IMPLEMENT_GPU_FUNCTIONS( BOOST_PP_TUPLE_ELEM(2,0,BASEINT_SIZE_PAIR), BOOST_PP_TUPLE_ELEM(2,1,BASEINT_SIZE_PAIR) )

BOOST_PP_SEQ_FOR_EACH(VLI_IMPLEMENT_GPU_FUNCTIONS_FOR, _, VLI_COMPILE_BASEINT_SIZE_PAIRS_SEQ)

#undef VLI_IMPLEMENT_GPU_FUNCTIONS_FOR
#undef VLI_IMPLEMENT_GPU_FUNCTIONS

} // namespace detail
} // namespace vli

