#include "hip/hip_runtime.h"
//
//  kernels_gpu_interface.h
//  vli
//
//  Created by Timothée Ewart on 22/08/11.
//  Copyright 2011 Université de Genève. All rights reserved.
//

#include "vli/detail/kernels_cpu_gpu.hpp"
#include "vli/detail/kernels_gpu.h"


#include "utils/cuPrintf.cu"

namespace vli {
    namespace detail {
    
/**
* local declaration of the commun kernels
*/

    template <typename BaseInt, std::size_t Size> 
    __host__ __device__ void kernels_addition_classic(BaseInt* x, BaseInt const* y);

    template <typename BaseInt, std::size_t Size>
    __host__ __device__ void kernel_negate_device(BaseInt* x);
 
    template <typename BaseInt>
    __host__ __device__ void kernels_addition_block(BaseInt* x, BaseInt const* y); 

    template <typename BaseInt, std::size_t Size>
    __host__ __device__ void kernels_multiplication_classic_truncate(BaseInt * res, BaseInt const* x, BaseInt const* y);	

    template <typename BaseInt>
    __host__ __device__ void kernels_multiplication_block(BaseInt const* x, BaseInt const* y, BaseInt* r);
 
    template <typename BaseInt>
    __host__ __device__ void kernels_multiplication_block_down(BaseInt const* x, BaseInt const*  y, BaseInt* r);

    template <typename BaseInt>
    __host__ __device__ void kernels_multiplication_block_up(BaseInt const* x, BaseInt const*  y, BaseInt* r);	

    template <typename BaseInt>
    __host__ __device__ void kernels_multiplication_base_reshaping(BaseInt const* x, BaseInt  const*  y, BaseInt* r);	

    template <typename BaseInt, std::size_t Size>
    __device__ void single_multiplication_device(BaseInt const* x, BaseInt const* y, BaseInt* z);  

    template <typename BaseInt, std::size_t Size>
    __device__ void polynome_polynome_multiplication_device(BaseInt const* p1, BaseInt const* p2, BaseInt* res);

// functions for the blocks algos

    template <typename BaseInt, std::size_t Size>
    __device__ void algo_triangle_up(int block_ai, int block_bj,unsigned int Order, BaseInt const* a,  BaseInt const* b, BaseInt* c);

    template <typename BaseInt, std::size_t Size>
    __device__ void algo_triangle_down(int block_ai, int block_bj,unsigned int Order, BaseInt const* a,  BaseInt const* b, BaseInt* c);
        
    template <typename BaseInt, std::size_t Size>
    __device__ void algo_diag(int block_ai, int block_bj,unsigned int Order, BaseInt const* a, BaseInt const* b, BaseInt* c);
    
    template <typename BaseInt, std::size_t Size>
    __device__ void algo_block_algo(int i, int j,unsigned int Order, BaseInt const* a, BaseInt const* b, BaseInt* c);

// functions for the diags algo

    template <typename BaseInt, std::size_t Size>
    __device__ void algo_diag_up(unsigned int i,unsigned int Order, BaseInt const* a,  BaseInt const* b, BaseInt* c);

    template <typename BaseInt, std::size_t Size>
    __device__ void algo_diag_down(unsigned int i,unsigned int Order, BaseInt const* a,  BaseInt const* b, BaseInt* c);
        
/**
* a kind of hook function with a little bit of arithmetic in case of signed int (multiplication)
*/

/**
* VLI_GPU functions
*/

template <typename BaseInt, std::size_t Size>
__device__  void single_multiplication_device(BaseInt const* x, BaseInt const* y, BaseInt* z)  
{
 //   int na(1),nb(1);

    bool result_is_negative = static_cast<bool>((x[Size-1] ^ y[Size-1]) >> data_bits<BaseInt>::value);
    if(result_is_negative)// test if 
    {
        kernel_negate_device<BaseInt,Size>(const_cast<BaseInt* >(x));
        kernels_multiplication_classic_truncate<BaseInt,Size>(z,x,y); 
        kernel_negate_device<BaseInt,Size>(const_cast<BaseInt* >(x));
    }
    else
    {
        kernels_multiplication_classic_truncate<BaseInt,Size>(z,x,y); 
    }
}
/**
I try to implement this stuff into the commun kernel impossible ... (3 days of trying)
the compiler does not want, we should move it inside ! 
**/
template <typename BaseInt, std::size_t Size>
__device__ void kernel_negate_device(BaseInt* x)
{
    BaseInt one(1);
    for(std::size_t i=0; i < Size-1; ++i)
        *(x+i) = (~*(x+i))&data_mask<BaseInt>::value;
    *(x+Size-1) = (~*(x+Size-1))&(base<BaseInt>::value+data_mask<BaseInt>::value);
    
    // TODO BUG!!!!
    kernels_addition_block(x,&one);
}
    
template <typename BaseInt, std::size_t Size>
__device__ void polynome_polynome_multiplication_device(unsigned int max_order, BaseInt const* p1, BaseInt const* p2, BaseInt* res)
{
    for(std::size_t je1 = 0; je1 < max_order; ++je1)
    {               
        for(std::size_t je2 = 0; je2 < max_order; ++je2)
        {
            for(std::size_t he1 = 0; he1 < max_order; ++he1)
            {                
                for(std::size_t he2 = 0; he2 < max_order; ++he2)
                {            
                    BaseInt inter[Size];
                    #pragma unroll
                    for(std::size_t i=0 ; i < Size ;++i)
                        inter[i] = 0;
        
                    std::size_t offset0 = ((je1+je2)*2*max_order + he1+he2)*Size;
                    std::size_t offset1 = (je1*max_order+he1)*Size;
                    std::size_t offset2 = (je2*max_order+he2)*Size;
                    single_multiplication_device<BaseInt,Size>(&p1[offset1],&p2[offset2],&inter[0]);
                    kernels_addition_classic<BaseInt,Size>(&res[offset0],&inter[0]);
                } 
            }
        }
    }
} 
        
/** n threads block algo **/
        
template <typename BaseInt, std::size_t Size>
void algo_triangle_up(int block_ai, int block_bj, unsigned int Order, BaseInt const* a,  BaseInt const* b, BaseInt *c){
    std::size_t n(0);
    int offset_block_ai = block_ai*Order;
    int offset_block_bj = block_bj*Order;            
    int offset_a, offset_b, offset_c;
    
    for(int i = 0; i < Order-1; ++i){
        for(int j = n; j < Order-1; ++j){
            BaseInt inter[Size];
            #pragma unroll
            for(std::size_t k=0 ; k < Size ;++k)
                inter[k] = 0;
            
            offset_a = (j-n+offset_block_ai)*Size;
            offset_b = (n+offset_block_bj)*Size;
            offset_c = ((offset_block_ai+offset_block_bj)*2+j)*Size;
             
            single_multiplication_device<BaseInt,Size>(&a[offset_a],&b[offset_b],inter);
            kernels_addition_classic<BaseInt,Size>(&c[offset_c],inter);
        }    
        n++;
    }        
}

template <typename BaseInt, std::size_t Size>
void algo_diag(int block_ai, int block_bj,  unsigned int Order, BaseInt const* a, BaseInt const* b, BaseInt *c){
    int OrderMinusOne = Order-1;
    int offset_block_ai = block_ai*Order;
    int offset_block_bj = block_bj*Order;   
    int offset_a, offset_b, offset_c;
    
    for(int i = 0; i < Order; ++i){
        BaseInt inter[Size];
        #pragma unroll
        for(std::size_t k=0 ; k < Size ;++k)
            inter[k] = 0;
        
        offset_a = (offset_block_bj+Order-1-i)*Size;
        offset_b = (offset_block_ai+i)*Size;
        offset_c = ((offset_block_ai+offset_block_bj)*2+OrderMinusOne)*Size;

        single_multiplication_device<BaseInt,Size>(&a[offset_a],&b[offset_b],inter);
        kernels_addition_classic<BaseInt,Size>(&c[offset_c],inter);
    }
    
}

template <typename BaseInt, std::size_t Size>
void algo_triangle_down(int block_ai, int block_bj, unsigned int Order, BaseInt const* a, BaseInt const* b, BaseInt *c){
    int n(0);
    int offset_block_ai = (block_ai+1)*(Order)-1;
    int offset_block_bj = (block_bj+1)*(Order)-1;
    int offset_a, offset_b, offset_c;
    
    for(int i = 0; i < Order-1; ++i){
        for(int j = n; j < Order-1; ++j){
            BaseInt inter[Size];
            #pragma unroll
            for(std::size_t k=0 ; k < Size ;++k)
                inter[k] = 0;
            
            offset_a = (offset_block_ai+n-j)*Size;
            offset_b = (offset_block_bj-n)*Size;
            offset_c = ((offset_block_ai+offset_block_bj)*2-2*Order+2-j)*Size;
            
            single_multiplication_device<BaseInt,Size>(&a[offset_a],&b[offset_b],&inter[0]);
            kernels_addition_classic<BaseInt,Size>(&c[offset_c],&inter[0]);
        }    
        n++;
    }         
}

template <typename BaseInt, std::size_t Size>
void algo_block_algo(int i, int j, unsigned int Order, BaseInt const* a, BaseInt const* b, BaseInt *c)
{
    algo_triangle_up<BaseInt,Size>(i,j,Order,a,b,c);
    algo_diag<BaseInt,Size>(i,j,Order,a,b,c);
    algo_triangle_down<BaseInt,Size>(i,j,Order,a,b,c);
}

/** n*n threads diag algo **/

template <typename BaseInt, std::size_t Size>
void algo_diag_up(unsigned int n, unsigned int Order, BaseInt const* a, BaseInt const* b, BaseInt *c)
{
    int qa,ra,qb,rb,pos; // find all indexes
    int offset_a, offset_b, offset_c;

    for(int i(0); i <= n; i++){
        BaseInt inter[Size];
        #pragma unroll
        for(std::size_t k=0 ; k < Size ;++k)
            inter[k] = 0;

        qa = i/Order;
        ra = i%Order;
        qb = (n-i)/Order;
        rb = (n-i)%Order;
        pos = 2*(qa+qb)*Order + (ra+rb);

        offset_a = (n-i)*Size;
        offset_b = i*Size;
        offset_c = pos*Size;
        
        single_multiplication_device<BaseInt,Size>(&a[offset_a],&b[offset_b],&inter[0]);
        kernels_addition_classic<BaseInt,Size>(&c[offset_c],&inter[0]);
//     std::cout << " qa " << qa << " ra " << ra << " qb " << qb << " rb " << rb << " pos " << pos << std::endl;  
//        result.coeffs_[pos] += p1.coeffs_[n-i]*p2.coeffs_[i];  
    }
}

template <typename BaseInt, std::size_t Size>
void algo_diag_down(unsigned int n, unsigned int Order, BaseInt const* a, BaseInt const* b, BaseInt *c)
{
    int qa,ra,qb,rb,pos; // find all indexes
    int offset_a, offset_b, offset_c;

    int j = Order*Order-1;
 
    for(int i(Order*Order-n+1); i < Order*Order; i++){
        BaseInt inter[Size];
        #pragma unroll
        for(std::size_t k=0 ; k < Size ;++k)
            inter[k] = 0;

        qa = i/Order;
        ra = i%Order;
        qb = j/Order;
        rb = j%Order;
        pos = 2*(qa+qb)*Order + (ra+rb);

        offset_a = j*Size;
        offset_b = i*Size;
        offset_c = pos*Size;
        
        single_multiplication_device<BaseInt,Size>(&a[offset_a],&b[offset_b],&inter[0]);
        kernels_addition_classic<BaseInt,Size>(&c[offset_c],&inter[0]);
        //std::cout << " qa " << qa << " ra " << ra << " qb " << qb << " rb " << rb << " pos " << pos << std::endl;   
        //result.coeffs_[pos] += p1.coeffs_[j]*p2.coeffs_[i];  
        j--;        
    }    
}

/**
* VLI_GPU_VECTOR functions
*/    
template  <typename BaseInt, std::size_t Size>
__global__ void inner_prod_vector(unsigned int max_order, std::size_t vector_size, BaseInt const* v1, BaseInt const* v2, BaseInt* res)
{
    unsigned int xIndex = blockIdx.x*blockDim.x + threadIdx.x; // all index on x // get poly one by one
    const std::size_t size_multiplicant = Size*max_order*max_order;
    const std::size_t size_product = Size*2*max_order*2*max_order;
    if(xIndex < vector_size){
        //multiplication between polynomial
        std::size_t offset_m = xIndex*size_multiplicant;
        std::size_t offset_p = xIndex*size_product;
        polynome_polynome_multiplication_device<BaseInt,Size>(max_order,&v1[offset_m],&v2[offset_m],&res[offset_p]); 
    }
}
    
    
template  <typename BaseInt, std::size_t Size>
__global__ void reduction_polynome(unsigned int max_order, std::size_t vector_size, BaseInt* v1)
{ 
    std::size_t size_poly = Size*max_order*max_order;
    for(std::size_t i=1 ; i < vector_size ; ++i){
        for(std::size_t j=0 ; j < max_order*max_order; ++j){ //additional loop
            std::size_t offset0 = j*Size;
            std::size_t offset1 = i*size_poly+j*Size;
            kernels_addition_classic<BaseInt,Size>(&v1[offset0],&v1[offset1]);
        }
    }
}
    
/**
* New algo based on block decomposition 
*/
    
template <typename BaseInt, std::size_t Size>
__global__ void inner_prod_vector_blocks(unsigned int Order, std::size_t vector_size, BaseInt const* A, BaseInt const* B, BaseInt* C)
{
    // remove the loops  
    unsigned int xIndex = blockIdx.x*blockDim.x + threadIdx.x; // all index on x // get poly one by one
    unsigned int yindex = threadIdx.y; // thread for the triangle/diag decomposition
    const std::size_t size_multiplicant = Size*Order*Order;
    const std::size_t size_product = Size*2*Order*2*Order;
    //multiplication between polynomial
    std::size_t offset_m = xIndex*size_multiplicant;
    std::size_t offset_p = xIndex*size_product;
    if(xIndex < vector_size){
        // first pass, half top right corner, 
        for(int j=0; j<=yindex; ++j)
            algo_block_algo<BaseInt, Size>(j,yindex-j,Order,&A[offset_m],&B[offset_m],&C[offset_p]);
        
        //second pass, half bottom left corner
        for(int j=yindex+1; j<Order; ++j)
            algo_block_algo<BaseInt, Size>(j,Order-j+yindex,Order,&A[offset_m],&B[offset_m],&C[offset_p]);          
    }
}
  

/**
* New algo based on diagonal decomposition 
*/

template <typename BaseInt, std::size_t Size>
__global__ void inner_prod_vector_diag(unsigned int Order, std::size_t vector_size, BaseInt const* A, BaseInt const* B, BaseInt* C)
{
    // remove the loops  
    unsigned int xIndex = blockIdx.x*blockDim.x + threadIdx.x; // all index on x // get poly one by one
    unsigned int yIndex = threadIdx.y; // thread for the triangle/diag decomposition
    const std::size_t size_multiplicant = Size*Order*Order;
    const std::size_t size_product = Size*2*Order*2*Order;
    //multiplication between polynomial
    std::size_t offset_m = xIndex*size_multiplicant;
    std::size_t offset_p = xIndex*size_product;
    if(xIndex < vector_size){
        //first pass
        algo_diag_up<BaseInt,Size>(yIndex                ,Order,&A[offset_m],&B[offset_m],&C[offset_p]);
        //second pass    
        algo_diag_down<BaseInt,Size>(Order*Order - yIndex,Order,&A[offset_m],&B[offset_m],&C[offset_p]); 
    }
}
/**
  * The C++ functions that call the kernels
  */
    
template <typename BaseInt, std::size_t Size>
void inner_product_vector(unsigned int Order, std::size_t vector_size, BaseInt const* A, BaseInt const* B, BaseInt* C, std::size_t threads_per_block) 
{
    std::size_t blocks_per_grid = vector_size/threads_per_block+1;
    dim3 dimgrid(blocks_per_grid,1,1);
    dim3 dimblock(threads_per_block,1,1);
    inner_prod_vector<BaseInt, Size> <<< dimgrid, dimblock >>>(Order, vector_size, A, B, C);
}
 
template <typename BaseInt, std::size_t Size>
void inner_product_vector_blocks(unsigned int Order, std::size_t vector_size, BaseInt const* A, BaseInt const* B, BaseInt *C)
{
/*
    std::size_t threads_per_block=1;
    std::size_t blocks_per_grid_x = vector_size/threads_per_block+1;
    dim3 dimgrid(blocks_per_grid_x,1,1);
    dim3 dimblock(threads_per_block,Order,1);
*/
  dim3 dimgrid(vector_size,1,1);
  dim3 dimblock(1,Order*Order,1);

   //inner_prod_vector_blocks<BaseInt,Size><<<dimgrid,dimblock>>>(Order,vector_size,A,B,C);      // nthreads version 
   inner_prod_vector_diag<BaseInt,Size><<<dimgrid,dimblock>>>(Order,vector_size,A,B,C);      // nthreads*nthreads version 
}
    
template <typename BaseInt, std::size_t Size>
void vector_reduction_inplace(unsigned int max_order, std::size_t vector_size, BaseInt* A)
{
    //the reduction should be // if executed on one smp
    dim3 dimgrid(1,1,1);
    dim3 dimblock(1,1,1);
    reduction_polynome<BaseInt, Size> <<< dimgrid, dimblock >>>(max_order, vector_size, A);
}        

#define VLI_IMPLEMENT_GPU_FUNCTIONS(TYPE, VLI_SIZE) \
    void inner_product_vector(vli_size_tag<VLI_SIZE>, unsigned int max_order, std::size_t vector_size, TYPE const* A, TYPE const* B, TYPE* C, std::size_t threads_per_block) \
        {inner_product_vector<TYPE,VLI_SIZE>(max_order,vector_size,A,B,C,threads_per_block);} \
    void vector_reduction_inplace(vli_size_tag<VLI_SIZE>, unsigned int max_order, std::size_t vector_size, TYPE* A) \
        {vector_reduction_inplace<TYPE,VLI_SIZE>(max_order,vector_size,A);} \
    void inner_product_vector_blocks(vli_size_tag<VLI_SIZE>, unsigned int Order, std::size_t vector_size, TYPE const* A, TYPE const* B, TYPE* C) \
        {inner_product_vector_blocks<TYPE,VLI_SIZE>(Order, vector_size, A, B, C);} 

    
#define VLI_IMPLEMENT_GPU_FUNCTIONS_FOR(r, data, BASEINT_SIZE_PAIR) \
    VLI_IMPLEMENT_GPU_FUNCTIONS( BOOST_PP_TUPLE_ELEM(2,0,BASEINT_SIZE_PAIR), BOOST_PP_TUPLE_ELEM(2,1,BASEINT_SIZE_PAIR) )

BOOST_PP_SEQ_FOR_EACH(VLI_IMPLEMENT_GPU_FUNCTIONS_FOR, _, VLI_COMPILE_BASEINT_SIZE_PAIRS_SEQ)

#undef VLI_IMPLEMENT_GPU_FUNCTIONS_FOR
#undef VLI_IMPLEMENT_GPU_FUNCTIONS

} // namespace detail
} // namespace vli

