#include "hip/hip_runtime.h"
//
//  kernels_gpu_interface.h
//  vli
//
//  Created by Timothée Ewart on 22/08/11.
//  Copyright 2011 Université de Genève. All rights reserved.
//

#include "vli/detail/kernels_cpu_gpu.hpp"
#include "vli/detail/kernels_gpu.h"


namespace vli {
namespace detail {

/**
* local declaration of the commun kernels
*/

template <typename BaseInt, std::size_t Size>
__host__ __device__ void kernels_addition_classic(BaseInt* x, BaseInt const* y);

template <typename BaseInt, std::size_t Size>
__host__ __device__ void kernel_negate_device(BaseInt* x);
 
template <typename BaseInt>
__host__ __device__ void kernels_addition_block(BaseInt* x, BaseInt const* y); 

template <typename BaseInt, std::size_t Size>
__host__ __device__ void kernels_addition_int(BaseInt* x, int a);

template <typename BaseInt, std::size_t Size>
__host__ __device__ void kernels_multiplication_classic_truncate(BaseInt * res, BaseInt const* x, BaseInt const* y);	

template <typename BaseInt>
__host__ __device__ void kernels_multiplication_block(BaseInt const* x, BaseInt const* y, BaseInt* r);

template <typename BaseInt>
__host__ __device__ void kernels_multiplication_block_down(BaseInt const* x, BaseInt const*  y, BaseInt* r);

template <typename BaseInt>
__host__ __device__ void kernels_multiplication_block_up(BaseInt const* x, BaseInt const*  y, BaseInt* r);	

template <typename BaseInt>
__host__ __device__ void kernels_multiplication_base_reshaping(BaseInt const* x, BaseInt  const*  y, BaseInt* r);	

template <typename BaseInt, std::size_t Size>
__host__ __device__ void kernels_multiplication_number(BaseInt* x, BaseInt a);

template <typename BaseInt, std::size_t Size>
__device__ void single_multiplication_device(BaseInt const* x, BaseInt const* y, BaseInt* z);  

template <typename BaseInt, std::size_t Size>
__device__ void polynome_polynome_multiplication_device(BaseInt const* p1, BaseInt const* p2, BaseInt* res);


/**
* a kind of hook function with a little bit of arithmetic in case of signed int (multiplication)
*/

/**
* VLI_GPU functions
*/

template <typename BaseInt, std::size_t Size>
__device__  void single_multiplication_device(BaseInt const* x, BaseInt const* y, BaseInt* z)  
{
 //   int na(1),nb(1);

    bool result_is_negative = static_cast<bool>((x[Size-1] ^ y[Size-1]) >> data_bits<BaseInt>::value);
    if(result_is_negative)// test if 
    {
        kernel_negate_device<BaseInt,Size>(const_cast<BaseInt* >(x));
        kernels_multiplication_classic_truncate<BaseInt,Size>(z,x,y); 
        kernel_negate_device<BaseInt,Size>(const_cast<BaseInt* >(x));
    }
    else
    {
        kernels_multiplication_classic_truncate<BaseInt,Size>(z,x,y); 
    }

/*    
    if( static_cast<bool>((x[Size-1]) >> data_bits<BaseInt>::value)){
        kernel_negate_device<BaseInt,Size>(const_cast<BaseInt* >(x));
        na = -1;
    }

    if( static_cast<bool>((y[Size-1]) >> data_bits<BaseInt>::value)){
        kernel_negate_device<BaseInt,Size>(const_cast<BaseInt* >(y));
        nb = -1;
    }

    kernels_multiplication_classic_truncate<BaseInt,Size>(z,x,y);

    if(nb*na == -1)
        kernel_negate_device<BaseInt,Size>(z);
       
    if(na == -1){
        kernel_negate_device<BaseInt,Size>(const_cast<BaseInt* >(x));
    }

    if(nb == -1){
        kernel_negate_device<BaseInt,Size>(const_cast<BaseInt* >(y));
    }*/
}
/**
I try to implement this stuff into the commun kernel impossible ... (3 days of trying)
the compiler does not want, we should move it inside ! 
**/
template <typename BaseInt, std::size_t Size>
__device__ void kernel_negate_device(BaseInt* x)
{
    BaseInt one(1);
    for(std::size_t i=0; i < Size-1; ++i)
        *(x+i) = (~*(x+i))&data_mask<BaseInt>::value;
    *(x+Size-1) = (~*(x+Size-1))&(base<BaseInt>::value+data_mask<BaseInt>::value);
    
    // TODO BUG!!!!
    kernels_addition_block(x,&one);
}

/**
* VLI_GPU_MONOMIAL functions
*/

template <typename BaseInt, std::size_t Size> 
__global__ void monome_polynome_multiplication(unsigned int max_order, BaseInt const* p, BaseInt const* m, BaseInt* res, std::size_t j_exp, std::size_t h_exp)
{ 
    for(std::size_t je = 0; je < max_order-j_exp; ++je)
    {
        for(std::size_t he = 0; he < max_order-h_exp; ++he)
        {   
            std::size_t offset0 = (je*max_order+he)*Size;
            std::size_t offset1 = ((j_exp+je)*max_order+h_exp+he)*Size;
            single_multiplication_device<BaseInt,Size>(&p[offset0],&m[0],&res[offset1]);        
        }
    }

}  

/**
* VLI_GPU_POLYNOMIAL functions
*/

template <typename BaseInt, std::size_t Size>
__global__ void polynome_polynome_addition(BaseInt* x, BaseInt const* y) 
{   
    std::size_t xIndex = blockIdx.x*blockDim.x + threadIdx.x; // all index on x
	std::size_t offset = xIndex*Size;
    kernels_addition_classic<BaseInt,Size>(&x[offset],&y[offset]);    //1 see line 148
}
  

template <typename BaseInt, std::size_t Size>
__global__ void polynome_polynome_subtraction(BaseInt* x, BaseInt* y) 
{
    std::size_t xIndex = blockIdx.x*blockDim.x + threadIdx.x; // all index on x
	std::size_t offset = xIndex*Size;
    kernel_negate_device<BaseInt,Size>(&y[offset]);
    kernels_addition_classic<BaseInt,Size>(&x[offset],&y[offset]);
}


template <typename BaseInt, std::size_t Size>
__device__ void polynome_polynome_multiplication_device(unsigned int max_order, BaseInt const* p1, BaseInt const* p2, BaseInt* res)
{
    for(std::size_t je1 = 0; je1 < max_order; ++je1)
    {
        for(std::size_t je2 = 0; je2 < max_order - je1; ++je2)
        {
            for(std::size_t he1 = 0; he1 < max_order; ++he1)
            {
                for(std::size_t he2 = 0; he2 < max_order - he1; ++he2)
                {
                    BaseInt inter[Size];
                    #pragma unroll
                    for(std::size_t i=0 ; i < Size ;++i)
                        inter[i] = 0;
                    std::size_t offset0 = ((je1+je2)*max_order + he1+he2)*Size;
                    std::size_t offset1 = (je1*max_order+he1)*Size;
                    std::size_t offset2 = (je2*max_order+he2)*Size;
                    single_multiplication_device<BaseInt,Size>(&p1[offset1],&p2[offset2],&inter[0]);
                    kernels_addition_classic<BaseInt,Size>(&res[offset0],&inter[0]);
                } 
            }
        }      
    }
} 


template <typename BaseInt, std::size_t Size>
__global__ void polynome_polynome_multiplication(unsigned int max_order, BaseInt const* x, BaseInt const* y, BaseInt* res)
{
    polynome_polynome_multiplication_device<BaseInt, Size>(max_order,x,y,res);
}
/**
* VLI_GPU_VECTOR functions
*/

    
template  <typename BaseInt, std::size_t Size>
__global__ void inner_prod_vector(unsigned int max_order, std::size_t vector_size, BaseInt const* v1, BaseInt const* v2, BaseInt* res)
{
    unsigned int xIndex = blockIdx.x*blockDim.x + threadIdx.x; // all index on x
    std::size_t size_poly = Size*max_order*max_order;
    //multiplication between polynomial
    if(xIndex < vector_size)
    {
        std::size_t offset = xIndex*size_poly;
        polynome_polynome_multiplication_device<BaseInt,Size>(max_order,&v1[offset],&v2[offset],&res[offset]); 
    }
}
    
    
template  <typename BaseInt, std::size_t Size>
__global__ void reduction_polynome(unsigned int max_order, std::size_t vector_size, BaseInt* v1)
{ 
    std::size_t size_poly = Size*max_order*max_order;
    for(std::size_t i=1 ; i < vector_size ; ++i){
        for(std::size_t j=0 ; j < max_order*max_order; ++j){ //additional loop
            std::size_t offset0 = j*Size;
            std::size_t offset1 = i*size_poly+j*Size;
            kernels_addition_classic<BaseInt,Size>(&v1[offset0],&v1[offset1]);
        }
    }
}
   
template <typename BaseInt, std::size_t Size>
__global__ void polynome_int_addition(BaseInt* x, int y)
{
    kernels_addition_int<BaseInt,Size>(x,y);
}

/**
  * The C++ functions that call the kernels
  */
/**
* VLI_GPU_POLYNOMIAL functions
*/
template <typename BaseInt, std::size_t Size>
void poly_mono_multiply(unsigned int max_order, BaseInt const* A, BaseInt const* B, BaseInt* C, std::size_t j_exp, std::size_t h_exp)
{
    dim3 dimgrid(1,1,1);
    dim3 dimblock(max_order*max_order,1,1);
    // TODO size_poly
    monome_polynome_multiplication<BaseInt, Size>  <<< dimgrid, dimblock >>>(max_order, A, B, C, j_exp, h_exp);
}

template <typename BaseInt, std::size_t Size>
void plus_assign_poly_int(BaseInt* A, int a)
{
    // TODO is 'int a' ok?
    dim3 dimgrid(1,1,1);
    dim3 dimblock(1,1,1);
    polynome_int_addition<BaseInt, Size> <<< dimgrid, dimblock>>>(A, a);
}

template <typename BaseInt, std::size_t Size>
void plus_assign_poly(unsigned int max_order, BaseInt* A, BaseInt const* B)
{
    dim3 dimgrid(1,1,1);
	dim3 dimblock(max_order*max_order,1,1);
	polynome_polynome_addition<BaseInt, Size> <<< dimgrid, dimblock >>>(A, B);
}

template <typename BaseInt, std::size_t Size>
void minus_assign_poly_destructive(unsigned int max_order, BaseInt* A, BaseInt* B)
{
    dim3 dimgrid(1,1,1);
	dim3 dimblock(max_order*max_order,1,1);
	polynome_polynome_subtraction<BaseInt, Size> <<< dimgrid, dimblock >>>(A, B);
}

template <typename BaseInt, std::size_t Size>
void poly_poly_multiply(unsigned int max_order, BaseInt const* A, BaseInt const* B, BaseInt* C)
{
    dim3 dimgrid(1,1,1);
    dim3 dimblock(1,1,1);
    //TODO size_poly
    polynome_polynome_multiplication<BaseInt, Size> <<< dimgrid, dimblock >>>(max_order, A, B, C); 
}

/**
* VLI_GPU_VECTOR functions
*/

template <typename BaseInt, std::size_t Size>
void inner_product_vector(unsigned int max_order, std::size_t vector_size, BaseInt const* A, BaseInt const* B, BaseInt* C, std::size_t threads_per_block) 
{
    std::size_t blocks_per_grid = vector_size/threads_per_block+1;
    dim3 dimgrid(blocks_per_grid,1,1);
	dim3 dimblock(threads_per_block,1,1);
    inner_prod_vector<BaseInt, Size> <<< dimgrid, dimblock >>>(max_order, vector_size, A, B, C);
} 

template <typename BaseInt, std::size_t Size>
void vector_reduction_inplace(unsigned int max_order, std::size_t vector_size, BaseInt* A)
{
    //the reduction should be // if executed on one smp
    dim3 dimgrid(1,1,1);
    dim3 dimblock(1,1,1);
    reduction_polynome<BaseInt, Size> <<< dimgrid, dimblock >>>(max_order, vector_size, A);
}

#define VLI_IMPLEMENT_GPU_FUNCTIONS(TYPE, VLI_SIZE) \
    void poly_mono_multiply(vli_size_tag<VLI_SIZE>, unsigned int max_order, TYPE const* A, TYPE const* B, TYPE* C, std::size_t j_exp, std::size_t h_exp) \
        {poly_mono_multiply<TYPE,VLI_SIZE>(max_order,A,B,C,j_exp,h_exp);} \
    void plus_assign_poly_int(vli_size_tag<VLI_SIZE>, TYPE* A, int a) \
        {plus_assign_poly_int<TYPE,VLI_SIZE>(A,a);} \
    void plus_assign_poly(vli_size_tag<VLI_SIZE>, unsigned int max_order, TYPE* A, TYPE const* B) \
        {plus_assign_poly<TYPE,VLI_SIZE>(max_order,A,B);} \
    void minus_assign_poly_destructive(vli_size_tag<VLI_SIZE>, unsigned int max_order, TYPE* A, TYPE* B) \
        {minus_assign_poly_destructive<TYPE,VLI_SIZE>(max_order,A,B);} \
    void poly_poly_multiply(vli_size_tag<VLI_SIZE>, unsigned int max_order, TYPE const* A, TYPE const* B, TYPE* C) \
        {poly_poly_multiply<TYPE,VLI_SIZE>(max_order,A,B,C);} \
    void inner_product_vector(vli_size_tag<VLI_SIZE>, unsigned int max_order, std::size_t vector_size, TYPE const* A, TYPE const* B, TYPE* C, std::size_t threads_per_block) \
        {inner_product_vector<TYPE,VLI_SIZE>(max_order,vector_size,A,B,C,threads_per_block);} \
    void vector_reduction_inplace(vli_size_tag<VLI_SIZE>, unsigned int max_order, std::size_t vector_size, TYPE* A) \
        {vector_reduction_inplace<TYPE,VLI_SIZE>(max_order,vector_size,A);}

#define VLI_IMPLEMENT_GPU_FUNCTIONS_FOR(r, data, BASEINT_SIZE_PAIR) \
    VLI_IMPLEMENT_GPU_FUNCTIONS( BOOST_PP_TUPLE_ELEM(2,0,BASEINT_SIZE_PAIR), BOOST_PP_TUPLE_ELEM(2,1,BASEINT_SIZE_PAIR) )

BOOST_PP_SEQ_FOR_EACH(VLI_IMPLEMENT_GPU_FUNCTIONS_FOR, _, VLI_COMPILE_BASEINT_SIZE_PAIRS_SEQ)

#undef VLI_IMPLEMENT_GPU_FUNCTIONS_FOR
#undef VLI_IMPLEMENT_GPU_FUNCTIONS

} // namespace detail
} // namespace vli

